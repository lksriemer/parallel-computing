#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <numeric>
#include <random>
#include <vector>

#include <hipcub/hipcub.hpp>

#include "hipblas.h"
#include <hip/hip_runtime.h>

constexpr int block_size = 256;

__global__ void vectorScalarProduct(float *vector, float scalar,
                                    std::size_t n) {
  const auto gid = threadIdx.x + blockIdx.x * blockDim.x;

  if (gid >= n) {
    return;
  }

  vector[gid] *= scalar;
}

__global__ void gemv(float *m, float *input, float *output, std::size_t n) {
  using BlockReduce = hipcub::BlockReduce<float, block_size>;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  const auto gid = threadIdx.x + blockIdx.x * blockDim.x;
  const auto tid = threadIdx.x;

  const auto block_start = gid / block_size * block_size;
  const auto block_end = block_start + block_size - 1;

  if (gid >= n * n) {
    return;
  }

  const float v = m[gid] * input[gid % n];

  const auto row = gid / n;

  const auto start_row = block_start / n;
  const auto end_row = block_end / n;

  for (auto cur_row = start_row; cur_row <= end_row; ++cur_row) {
    const float sum = BlockReduce(temp_storage).Sum(row == cur_row ? v : .0);
    if (tid == 0) {
      atomicAdd(output + cur_row, sum);
    }
  }
}

__global__ void gemv_row_blocked(float *mat, float *input, float *output,
                                 std::size_t n, std::size_t m) {
  using BlockReduce = hipcub::BlockReduce<float, block_size>;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  const auto gid = threadIdx.x + blockIdx.x * blockDim.x;
  const auto tid = threadIdx.x;

  if (gid >= block_size * m) {
    return;
  }

  const auto row = gid / block_size;

  float sum = .0;
  for (auto i = tid; i < n; i += block_size) {
    sum += mat[row * n + i] * input[i];
  }

  // Return the warp-wide sum to lane0
  const float full_sum = BlockReduce(temp_storage).Sum(sum);
  if (tid == 0) {
    output[row] = full_sum;
  }
}

__global__ void gemv_row_blocked_d(double *mat, double *input, double *output,
                                   std::size_t n, std::size_t m) {
  using BlockReduce = hipcub::BlockReduce<double, block_size>;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  const auto gid = threadIdx.x + blockIdx.x * blockDim.x;
  const auto tid = threadIdx.x;

  if (gid >= block_size * m) {
    return;
  }

  const auto row = gid / block_size;

  double sum = .0;
  for (auto i = tid; i < n; i += block_size) {
    sum += mat[row * n + i] * input[i];
  }

  // Return the warp-wide sum to lane0
  const double full_sum = BlockReduce(temp_storage).Sum(sum);
  if (tid == 0) {
    output[row] = full_sum;
  }
}

void gen_rand_vec(float *v, int n) {
  std::uniform_real_distribution<float> distribution(-1000., 1000.);
  std::mt19937 engine;
  auto generator = std::bind(distribution, engine);
  std::generate_n(v, n, generator);
}

void gemv_cpu(float *mat, float *input, float *output, std::size_t n,
              std::size_t m) {
  for (auto row = 0; row < m; ++row) {
    for (auto col = 0; col < n; ++col) {
      output[row] += mat[row * n + col] * input[col];
    }
  }
}

int main_par_6(int argc, char *argv[]) {
  const std::size_t n = argc >= 2 ? std::stoi(argv[1]) : 10;
  const std::size_t m = argc >= 3 ? std::stoi(argv[2]) : n;

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  std::vector<float> h_input(n);
  std::iota(h_input.begin(), h_input.end(), 0);
  std::vector<float> h_output(n);
  std::fill(h_output.begin(), h_output.end(), 0);
  std::vector<float> h_matrix(n * n);
  std::iota(h_matrix.begin(), h_matrix.end(), 0);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float *d_input;
  hipMalloc(&d_input, n * sizeof(float));
  hipMemcpy(d_input, h_input.data(), n * sizeof(float),
             hipMemcpyHostToDevice);
  float *d_output;
  hipMalloc(&d_output, n * sizeof(float));
  hipMemcpy(d_output, h_output.data(), n * sizeof(float),
             hipMemcpyHostToDevice);
  float *d_matrix;
  hipMalloc(&d_matrix, n * n * sizeof(float));
  hipMemcpy(d_matrix, h_matrix.data(), n * n * sizeof(float),
             hipMemcpyHostToDevice);

  int blockSize = block_size;
  int gridSize = (m * block_size + blockSize - 1) / blockSize;

  float one = 1.;
  float zero = 0.;

  hipEventRecord(start);

  //   hipblasSgemv(handle, HIPBLAS_OP_T,
  //     m, n,
  //     &one,
  //     d_matrix, m,
  //     d_input, 1,
  //     &zero,
  //     d_output, 1);
  gemv_row_blocked<<<gridSize, blockSize>>>(d_matrix, d_input, d_output, n, m);

  hipEventRecord(stop);

  hipMemcpy(h_output.data(), d_output, n * sizeof(float),
             hipMemcpyDeviceToHost);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  //   for (int i = 0; i < n; ++i) {
  //     std::cout << h_output[i] << " ";
  //   }
  //   std::cout << std::endl;

  const float ops = n * n + 2 * n;
  std::cout << "took " << milliseconds << "ms which is "
            << (ops * sizeof(float)) / milliseconds * 1000. / 1000000000.
            << "GByte/s" << std::endl;

  return 0;
}

int main_par_7(int argc, char *argv[]) {
  const std::size_t n = argc >= 2 ? std::stoi(argv[1]) : 10;
  const std::size_t m = argc >= 3 ? std::stoi(argv[2]) : n;

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  std::vector<double> h_input(n);
  std::iota(h_input.begin(), h_input.end(), 0);
  std::vector<double> h_output(n);
  std::fill(h_output.begin(), h_output.end(), 0);
  std::vector<double> h_matrix(n * n);
  std::iota(h_matrix.begin(), h_matrix.end(), 0);

  double *d_input;
  hipMalloc(&d_input, n * sizeof(double));
  hipMemcpy(d_input, h_input.data(), n * sizeof(double),
             hipMemcpyHostToDevice);
  double *d_output;
  hipMalloc(&d_output, n * sizeof(double));
  hipMemcpy(d_output, h_output.data(), n * sizeof(double),
             hipMemcpyHostToDevice);
  double *d_matrix;
  hipMalloc(&d_matrix, n * n * sizeof(double));
  hipMemcpy(d_matrix, h_matrix.data(), n * n * sizeof(double),
             hipMemcpyHostToDevice);

  const auto min_iters = 100;
  const auto min_seconds = 5;
  auto total_runtime = 0.;
  auto i = 0;
  for (; i < min_iters ||
         (i >= min_iters && total_runtime < min_seconds * 1000000);
       ++i) {

    // Compute y = Ax
    int blockSize = block_size;
    int gridSize = (m * block_size + blockSize - 1) / blockSize;

    double one = 1.;
    double zero = 0.;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

      hipblasDgemv(handle, HIPBLAS_OP_T,
        m, n,
        &one,
        d_matrix, m,
        d_input, 1,
        &zero,
        d_output, 1);
    // gemv_row_blocked_d<<<gridSize, blockSize>>>(d_matrix, d_input, d_output, n,
    //                                           m);

    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    const auto elapsed = (int)(milliseconds * 1000);
    total_runtime += elapsed;
  }

  const auto avg_runtime = total_runtime / i;

  std::cout << "avg runtime: " << avg_runtime << " at size " << n << std::endl;

  return 0;
}

int main(int argc, char *argv[]) { return main_par_7(argc, argv); }